#include "hip/hip_runtime.h"
#include "misc_kernels_impl.cuh"
#include "misc_kernels.h"
#include "dispatch_utils.h"

Tensor add(Tensor a, Tensor b) {
    assert(a.shape.dataExtent == b.shape.dataExtent);
    assert(a.dtype() == b.dtype());
    assert(a.is_contiguous());
    assert(b.is_contiguous());

    int threadsPerBlock = 1024;
    int blocksPerGrid = (a.numel() + threadsPerBlock - 1) / threadsPerBlock;

    auto stream = getCurrentCUDAStream();

    Tensor out = Tensor::empty_like(a);

    dispatch(out.scalar_type(), [&]<typename scalar_t>() {
        add_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            a.data_ptr<scalar_t>(), b.data_ptr<scalar_t>(), out.data_ptr<scalar_t>(), out.numel());
    });

    return out;
}

void mul_add(Tensor x, Tensor scale, Tensor bias) {
    // assert(scale.shape.data == bias.shape.data);
    // FIXME FIXME
    assert(x.numel() % scale.numel() == 0);
    assert(x.numel() % bias.numel() == 0);
    assert(x.dtype() == scale.dtype());
    assert(x.dtype() == bias.dtype());

    constexpr int unroll = 8;

    assert((uintptr_t)x.data_ptr() % (x.scalar_size() * unroll) == 0);
    assert((uintptr_t)scale.data_ptr() % (x.scalar_size() * unroll) == 0);
    assert((uintptr_t)bias.data_ptr() % (x.scalar_size() * unroll) == 0);

    assert(x.numel() % unroll == 0);
    assert(scale.numel() % unroll == 0);
    assert(bias.numel() % unroll == 0);

    int threadsPerBlock = 1024;
    int blocksPerGrid = (x.numel() + threadsPerBlock * unroll - 1) / (threadsPerBlock * unroll);

    auto stream = getCurrentCUDAStream();

    dispatch(x.scalar_type(), [&]<typename scalar_t>() {
        mul_add_kernel<scalar_t, unroll><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            x.data_ptr<scalar_t>(), scale.data_ptr<scalar_t>(), bias.data_ptr<scalar_t>(), x.numel(), scale.numel(), bias.numel());
    });
}

Tensor embedding(Tensor input_id, Tensor lookup) {
    assert(input_id.dtype() == Tensor::INT32);
    assert(lookup.ndims() == 2);

    auto shapeOut = input_id.shape;
    shapeOut.dataExtent.push_back(lookup.shape[-1]);

    auto stream = getCurrentCUDAStream();

    Tensor out = Tensor::empty(shapeOut, lookup.scalar_type(), input_id.device());

    dispatch(out.scalar_type(), [&]<typename scalar_t>() {
        EmbeddingKernel<<<input_id.numel(), std::min(lookup.shape[-1], 1024), 0, stream>>>(
            input_id.data_ptr<int32_t>(), out.data_ptr<scalar_t>(), lookup.data_ptr<scalar_t>(), lookup.shape[-1]);
    });

    return out;
}

Tensor argmax_sample(Tensor logits) {
    assert(logits.ndims() == 2);

    auto stream = getCurrentCUDAStream();

    Tensor out = Tensor::empty({logits.shape[0]}, Tensor::INT32, logits.device());

    dispatch(logits.scalar_type(), [&]<typename scalar_t>() {
        argmax_sample_kernel<<<logits.shape[0], std::min(logits.shape[1], 1024), 0, stream>>>(
            logits.data_ptr<scalar_t>(), out.data_ptr<int32_t>(), logits.shape[1]
        );
    });

    return out;
}

void splitqkv(Tensor qkv, Tensor q, Tensor k, Tensor v) {
    // FIXME FIXME
    // assert(qkv.shape[0] == q.shape[0]);
    // assert(qkv.shape[0] == k.shape[0]);
    // assert(qkv.shape[0] == v.shape[0]);

    auto stream = getCurrentCUDAStream();

    int dim_q = q.shape[-1] * q.shape[-2];
    int dim_k = k.shape[-1] * k.shape[-2];
    int dim_v = v.shape[-1] * v.shape[-2];

    assert(dim_k == dim_v);
    assert(dim_q + dim_k + dim_v == qkv.shape[-1]);
    
    int num_tokens = qkv.numel() / qkv.shape[-1];

    dispatch(qkv.scalar_type(), [&]<typename scalar_t>() {
        splitqkv_kernel<<<num_tokens, std::min(qkv.shape[-1], 1024), 0, stream>>>(
            qkv.data_ptr<scalar_t>(),
            q.data_ptr<scalar_t>(),
            k.data_ptr<scalar_t>(),
            v.data_ptr<scalar_t>(),
            dim_q,
            dim_k
        );
    });

}

template<size_t N>
std::array<Tensor, N> split_mod(Tensor input) {
    assert(input.shape[-1] % N == 0);

    int threadsPerBlock = 1024;
    int blocksPerGrid = (input.numel() + threadsPerBlock - 1) / threadsPerBlock;

    auto stream = getCurrentCUDAStream();

    auto shapeOut = input.shape;
    shapeOut[-1] /= N;

    std::array<Tensor, N> out;
    for (int k = 0; k < N; k++) {
        out[k] = Tensor::empty(shapeOut, input.scalar_type(), input.device());
    }

    dispatch(input.scalar_type(), [&]<typename scalar_t>() {
        std::array<scalar_t *, N> outPtr;
        for (int k = 0; k < N; k++) {
            outPtr[k] = out[k].template data_ptr<scalar_t>();
        }
        split_mod_kernel<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            input.data_ptr<scalar_t>(),
            outPtr, input.numel());
    });

    return out;
}

Tensor quant_static(Tensor x, float scale) {
    Tensor out = Tensor::empty(x.shape, Tensor::INT8, x.device());

    constexpr int unroll = 8;

    assert((uintptr_t)x.data_ptr() % (x.scalar_size() * unroll) == 0);

    int threadsPerBlock = 1024;
    int blocksPerGrid = (x.numel() + threadsPerBlock * unroll - 1) / (threadsPerBlock * unroll);

    auto stream = getCurrentCUDAStream();

    dispatch(x.scalar_type(), [&]<typename scalar_t>() {
        quant_kernel_static<scalar_t, unroll><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            x.data_ptr<scalar_t>(), out.data_ptr<int8_t>(), (scalar_t)scale, x.numel());
    });

    return out;
}

Tensor quant_static_fuse_gelu(Tensor x, float scale) {
    Tensor out = Tensor::empty(x.shape, Tensor::INT8, x.device());

    constexpr int unroll = 8;

    assert((uintptr_t)x.data_ptr() % (x.scalar_size() * unroll) == 0);


    int threadsPerBlock = 1024;
    int blocksPerGrid = (x.numel() + threadsPerBlock * unroll - 1) / (threadsPerBlock * unroll);

    auto stream = getCurrentCUDAStream();

    dispatch(x.scalar_type(), [&]<typename scalar_t>() {
        quant_kernel_static_fuse_gelu<scalar_t, unroll><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
            x.data_ptr<scalar_t>(), out.data_ptr<int8_t>(), (scalar_t)scale, x.numel());
    });

    return out;
}

void cast(Tensor input, Tensor output) {
    assert(input.is_contiguous());
    assert(output.is_contiguous());
    assert(input.shape.dataExtent == output.shape.dataExtent);

    auto stream = getCurrentCUDAStream();

    dispatch(input.scalar_type(), [&]<typename input_t>() {
        dispatch(output.scalar_type(), [&]<typename output_t>() {
            constexpr int unroll = 16 / std::max(sizeof(input_t), sizeof(output_t));

            int threadsPerBlock = 1024;
            int blocksPerGrid = (int)ceilDiv<int64_t>(input.numel(), threadsPerBlock * unroll);

            cast_kernel<input_t, output_t, unroll><<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
                input.data_ptr<input_t>(), output.data_ptr<output_t>(), input.numel());

            checkCUDA(hipGetLastError());
        });
    });
}

Tensor topk(Tensor x, int k) {
    constexpr int MAXK = 64 + 4;

    const int N = x.shape[-1];
    const int batch = x.numel() / N;

    assert(k <= N);
    assert(k <= MAXK);

    auto outShape = x.shape;
    outShape[-1] = k;
    outShape.dataStride.clear();


    Tensor out = Tensor::empty(outShape, Tensor::INT32, x.device());

    auto stream = getCurrentCUDAStream();

    dispatchVal(k, std::make_integer_sequence<int, MAXK + 1>(), [&]<int K>() {
        if constexpr (K == 0) {
            assert(false);
            return;
        }
        if constexpr (K > 0) {
            dispatch(x.scalar_type(), [&]<typename scalar_t>() {
                topk_kernel<scalar_t, K><<<ceilDiv(batch, 32), 32, 0, stream>>>(
                    x.data_ptr<scalar_t>(),
                    out.data_ptr<int>(),
                    N, x.stride(-2), batch
                );
                checkCUDA(hipGetLastError());
            });
        }
    });

    return out;
}

template std::array<Tensor, 2> split_mod<2>(Tensor input);
template std::array<Tensor, 3> split_mod<3>(Tensor input);
template std::array<Tensor, 4> split_mod<4>(Tensor input);
template std::array<Tensor, 5> split_mod<5>(Tensor input);
template std::array<Tensor, 6> split_mod<6>(Tensor input);