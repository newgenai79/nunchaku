#include "hip/hip_runtime.h"
#include "layernorm_kernels_impl.cuh"
#include "dispatch_utils.h"

void rms_norm(Tensor &out,    // [..., hidden_size]
              Tensor &input,  // [..., hidden_size]
              Tensor &weight, // [hidden_size]
              float epsilon,
              bool use_quant) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;
  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const hipStream_t stream = getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "rms_norm_kernel", [&] {
    if (use_quant) {
      vllm::rms_norm_kernel<scalar_t, int8_t, true><<<grid, block, 0, stream>>>(
        out.data_ptr<int8_t>(), input.data_ptr<scalar_t>(),
        weight.data_ptr<scalar_t>(), epsilon, num_tokens, hidden_size);
    } else {
      vllm::rms_norm_kernel<scalar_t, scalar_t, false><<<grid, block, 0, stream>>>(
        out.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(),
        weight.data_ptr<scalar_t>(), epsilon, num_tokens, hidden_size);
    }
  });
}

void layernorm_general(Tensor out, Tensor input, Tensor weight, Tensor bias, float epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;
  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 256));
  block.x = 32 * ((block.x + 31) / 32);

  size_t size_shmem = input.scalar_size() * hidden_size;
  
  const hipStream_t stream = getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "generalLayerNorm", [&] {
    using T = typename packed_as<scalar_t, 2>::type;
    vllm::generalLayerNorm<T, half, true><<<grid, block, size_shmem, stream>>>(
        reinterpret_cast<T*>(input.data_ptr<scalar_t>()), 
        weight.valid() ? reinterpret_cast<T*>(weight.data_ptr<scalar_t>()) : nullptr, 
        bias.valid() ? reinterpret_cast<T*>(bias.data_ptr<scalar_t>()) : nullptr,
        reinterpret_cast<T*>(out.data_ptr<scalar_t>()), 
        epsilon, num_tokens, hidden_size, nullptr, nullptr, nullptr, true
      );
  });
}

void rms_norm_general(Tensor &out,    // [..., hidden_size]
              Tensor &input,  // [..., hidden_size]
              Tensor &weight, // [hidden_size]
              Tensor &scaling, // [tokens] or [1]
              float epsilon,
              bool use_per_token_quant) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;
  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  block.x = 32 * ((block.x + 31) / 32);
  
  const hipStream_t stream = getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "generalLayerNorm", [&] {
    using T = scalar_t;
    if (use_per_token_quant) {
      // per-token
      vllm::generalLayerNorm<T, half><<<grid, block, 0, stream>>>(
        reinterpret_cast<T*>(input.data_ptr<scalar_t>()), 
        reinterpret_cast<T*>(weight.data_ptr<scalar_t>()), nullptr,
        nullptr, epsilon, num_tokens, hidden_size, nullptr, scaling.data_ptr<half>(),
        out.data_ptr<int8_t>(), false
      );
      // input, gamma, beta, normed_output, eps, tokens, hidden_dim, per_tensor_scale, per_token_scale
      // normed_output_quant, use_shmem
        // out.data_ptr<int8_t>(), input.data_ptr<scalar_t>(),
        // weight.data_ptr<scalar_t>(), epsilon, num_tokens, hidden_size);
    } else {
      // per-tensor
      vllm::generalLayerNorm<T, half><<<grid, block, 0, stream>>>(
        reinterpret_cast<T*>(input.data_ptr<scalar_t>()), 
        reinterpret_cast<T*>(weight.data_ptr<scalar_t>()), nullptr,
        nullptr, epsilon, num_tokens, hidden_size, scaling.data_ptr<half>(), nullptr,
        out.data_ptr<int8_t>(), false
      );
    }
  });
}

void rms_norm_general_fuse_sum(Tensor &out,    // [..., hidden_size]
              Tensor &input,  // [..., hidden_size]
              Tensor &weight, // [hidden_size]
              Tensor &input_sum, // [tokens] or [1]
              Tensor &scaling, // [tokens] or [1]
              float epsilon,
              bool use_per_token_quant) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;
  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  block.x = 32 * ((block.x + 31) / 32);
  
  const hipStream_t stream = getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "generalLayerNorm_fuse_sum", [&] {
    using T = scalar_t;
    if (use_per_token_quant) {
      // per-token
      vllm::generalLayerNorm_fuse_sum<T, half><<<grid, block, 0, stream>>>(
        reinterpret_cast<T*>(input.data_ptr<scalar_t>()), 
        reinterpret_cast<T*>(weight.data_ptr<scalar_t>()), nullptr,
        nullptr, epsilon, num_tokens, hidden_size, input_sum.data_ptr<half>(), nullptr, scaling.data_ptr<half>(),
        out.data_ptr<int8_t>(), false
      );
      // input, gamma, beta, normed_output, eps, tokens, hidden_dim, per_tensor_scale, per_token_scale
      // normed_output_quant, use_shmem
        // out.data_ptr<int8_t>(), input.data_ptr<scalar_t>(),
        // weight.data_ptr<scalar_t>(), epsilon, num_tokens, hidden_size);
    } else {
      // per-tensor
      // Rasing error here
      // Not implemented per-tensor input_sum
      assert(false);
      
      vllm::generalLayerNorm_fuse_sum<T, half><<<grid, block, 0, stream>>>(
        reinterpret_cast<T*>(input.data_ptr<scalar_t>()), 
        reinterpret_cast<T*>(weight.data_ptr<scalar_t>()), nullptr,
        nullptr, epsilon, num_tokens, hidden_size, nullptr, scaling.data_ptr<half>(), nullptr,
        out.data_ptr<int8_t>(), false
      );
    }
  });
}



void invoke_dequant_add_residual_rms_norm_quant(
    Tensor &out,      // [..., hidden_size]
    Tensor &input,    // [..., hidden_size]
    Tensor &residual, // [..., hidden_size]
    Tensor &gamma,    // [hidden_size]
    half scale,
    float epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;
  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));
  const hipStream_t stream = getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      residual.scalar_type(), "dequant_add_residual_rms_norm_quant_kernel",
      [&] {
          vllm::dequant_add_residual_rms_norm_quant_kernel<scalar_t, half, false>
            <<<grid, block, 0, stream>>>(
                input.data_ptr<int32_t>(), residual.data_ptr<scalar_t>(),
                out.data_ptr<int8_t>(), gamma.data_ptr<scalar_t>(), epsilon,
                scale, num_tokens, hidden_size);
      });
}

void invoke_dequant_add_residual_rms_norm_quant(
    Tensor &out,      // [..., hidden_size]
    Tensor &input,    // [..., hidden_size]
    Tensor &residual, // [..., hidden_size]
    Tensor &gamma,    // [hidden_size]
    Tensor &scale,    // [num_tokens]
    float epsilon) {
  int hidden_size = input.size(-1);
  int num_tokens = input.numel() / hidden_size;

  dim3 grid(num_tokens);
  dim3 block(std::min(hidden_size, 1024));

  const hipStream_t stream = getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      residual.scalar_type(), "dequant_add_residual_rms_norm_quant_kernel",
      [&] {
          vllm::dequant_add_residual_rms_norm_quant_kernel<scalar_t, half*, true>
            <<<grid, block, 0, stream>>>(
                input.data_ptr<int32_t>(), residual.data_ptr<scalar_t>(),
                out.data_ptr<int8_t>(), gamma.data_ptr<scalar_t>(), epsilon,
                scale.data_ptr<half>(), num_tokens, hidden_size);
      });
}
